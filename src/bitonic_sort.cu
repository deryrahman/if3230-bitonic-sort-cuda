
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>

struct timeval st, et;
__device__ __host__ void swap(int*, int, int);
void rng(int*, int);
int getMax(int*, int);
void buildDummy(int*, int, int, int);
__global__ void compareAndSwap(int*, int, int, int);
void impBitonicSortPar(int*, int, int);
void impBitonicSortSer(int*, int);
int getPowTwo(int);
void writeToFile(int*, int, char*);
bool isValid(int*, int*, int);

int main(int argc, char **argv) {
  int n, dummy_n, t = 512;

  if (argc < 2) {
    printf("Usage: %s <n> <p>\nwhere <n> is problem size, <p> is number of thread (optional)\n", argv[0]);
    exit(1);
  }

  if (argc == 3){
    t = atoi(argv[2]);
  }

  n = atoi(argv[1]);
  dummy_n = getPowTwo(n);

  int *arr, *arr_ser, *d_arr;

  arr = (int*) malloc(dummy_n*sizeof(int));
  arr_ser = (int*) malloc(dummy_n*sizeof(int));
  rng(arr,n);
  int max_x = getMax(arr,n);
  buildDummy(arr,n,dummy_n,max_x);
  memcpy(arr_ser, arr, dummy_n*sizeof(int));
  hipMalloc((void **)&d_arr, dummy_n*sizeof(int));
  hipMemcpy(d_arr, arr, dummy_n*sizeof(int), hipMemcpyHostToDevice);

  // write random numbers to input file
  writeToFile(arr,n,"./data/input");

  // execute paralel
  gettimeofday(&st,NULL);
  impBitonicSortPar(d_arr,dummy_n,t);
  gettimeofday(&et,NULL);
  int elapsed_paralel = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
  printf("Execution paralel time: %d micro sec\n",elapsed_paralel);

  // execute serial
  gettimeofday(&st,NULL);
  impBitonicSortSer(arr_ser,dummy_n);
  gettimeofday(&et,NULL);
  int elapsed_serial = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
  printf("Execution serial time: %d micro sec\n",elapsed_serial);

  // calculate speedup
  float speedup = (float)elapsed_serial/elapsed_paralel;
  printf("Speedup : %.3f\n",speedup);
  // calculate efficiency
  float eff = 100*speedup/t;
  printf("Efficiency : %.3f%\n",eff);

  hipMemcpy(arr, d_arr, dummy_n*sizeof(int), hipMemcpyDeviceToHost);

  // check test
  bool valid = isValid(arr_ser,arr,dummy_n);
  if(valid){
    printf("Test Passed\n");
  } else {
    printf("Test Failed\n");
  }
  writeToFile(arr,n,"./data/output");
  free(arr);
  free(arr_ser);
  hipFree(d_arr);
  return 0;
}

void writeToFile(int* arr, int n, char* path){
  FILE* f = fopen(path,"w");
  for(int i=0; i<n; i++) {
      fprintf(f, "%d\n", arr[i]);
  }
  fclose(f);
}

void rng(int* arr, int n) {
  int seed = 13515097;
  srand(seed);
  for(long i = 0; i < n; i++) {
      arr[i] = (int)rand();
  }
}

int getMax(int* arr, int n){
  int max_x = arr[0];
  for(int i=0; i<n; i++){
    max_x = ((max_x<arr[i])?arr[i]:max_x);
  }
  return max_x;
}

void buildDummy(int* arr,int N,int dummy_n, int max_x){
  for(long i = N; i < dummy_n; i++) {
    arr[i]=max_x;
  }
}

__device__ __host__ void swap(int* a, int i, int j) {
  int t;
  t = a[i];
  a[i] = a[j];
  a[j] = t;
}

__global__ void compareAndSwap(int* a, int n, int k, int j){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int ij=i^j;
  if ((ij)>i) {
    // monotonic increasing
    if ((i&k)==0 && a[i] > a[ij]) swap(a,i,ij);
    // monotonic decreasing
    if ((i&k)!=0 && a[i] < a[ij]) swap(a,i,ij);
  }
}

/*
Imperative paralel bitonic sort
*/
void impBitonicSortPar(int* a, int n, int t) {
  int j,k;
  int blocks = (n+t-1)/t;
  int threads = t;
  dim3 grid_dim(blocks,1);
  dim3 block_dim(threads,1);
  for (k=2; k<=n; k=2*k) {
    for (j=k>>1; j>0; j=j>>1) {
      compareAndSwap<<<grid_dim,block_dim>>>(a, n, k, j);
      hipDeviceSynchronize();
    }
  }
}

void impBitonicSortSer(int* a, int n){
  int i,j,k;

  for (k=2; k<=n; k=2*k) {
    for (j=k>>1; j>0; j=j>>1) {
      for (i=0; i<n; i++) {
        int ij=i^j;
        if ((ij)>i) {
          // monotonic increasing
          if ((i&k)==0 && a[i] > a[ij]) swap(a,i,ij);
          // monotonic decreasing
          if ((i&k)!=0 && a[i] < a[ij]) swap(a,i,ij);
        }
      }
    }
  }
}

int getPowTwo(int n){
  int d=1;
  while (d>0 && d<n) d<<=1;
  return d;
}

bool isValid(int* arr1, int* arr2, int n){
  for(int i=0; i<n; i++){
    if(arr1[i]!=arr2[i]) return 0;
  }
  return 1;
}
